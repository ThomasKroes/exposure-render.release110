#include "hip/hip_runtime.h"
/*
	Copyright (c) 2011, T. Kroes <t.kroes@tudelft.nl>
	All rights reserved.

	Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:

	- Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
	- Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other materials provided with the distribution.
	- Neither the name of the <ORGANIZATION> nor the names of its contributors may be used to endorse or promote products derived from this software without specific prior written permission.
	
	THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include "Core.cuh"

texture<short, hipTextureType3D, hipReadModeNormalizedFloat>		gTexDensity;
texture<short, hipTextureType3D, hipReadModeNormalizedFloat>		gTexGradientMagnitude;
texture<float, hipTextureType3D, hipReadModeElementType>			gTexExtinction;
texture<float, hipTextureType1D, hipReadModeElementType>			gTexOpacity;
texture<float4, hipTextureType1D, hipReadModeElementType>			gTexDiffuse;
texture<float4, hipTextureType1D, hipReadModeElementType>			gTexSpecular;
texture<float, hipTextureType1D, hipReadModeElementType>			gTexRoughness;
texture<float4, hipTextureType1D, hipReadModeElementType>			gTexEmission;
texture<uchar4, hipTextureType2D, hipReadModeNormalizedFloat>		gTexRunningEstimateRgba;

hipArray* gpDensityArray				= NULL;
hipArray* gpGradientMagnitudeArray		= NULL;
hipArray* gpOpacityArray				= NULL;
hipArray* gpDiffuseArray				= NULL;
hipArray* gpSpecularArray				= NULL;
hipArray* gpRoughnessArray				= NULL;
hipArray* gpEmissionArray				= NULL;

CD float3		gAaBbMin;
CD float3		gAaBbMax;
CD float3		gInvAaBbMin;
CD float3		gInvAaBbMax;
CD float		gIntensityMin;
CD float		gIntensityMax;
CD float		gIntensityRange;
CD float		gIntensityInvRange;
CD float		gStepSize;
CD float		gStepSizeShadow;
CD float		gDensityScale;
CD float		gGradientDelta;
CD float		gInvGradientDelta;
CD float3		gGradientDeltaX;
CD float3		gGradientDeltaY;
CD float3		gGradientDeltaZ;
CD int			gFilmWidth;
CD int			gFilmHeight;
CD int			gFilmNoPixels;
CD int			gFilterWidth;
CD float		gFilterWeights[10];
CD float		gExposure;
CD float		gInvExposure;
CD float		gGamma;
CD float		gInvGamma;
CD float		gDenoiseEnabled;
CD float		gDenoiseWindowRadius;
CD float		gDenoiseInvWindowArea;
CD float		gDenoiseNoise;
CD float		gDenoiseWeightThreshold;
CD float		gDenoiseLerpThreshold;
CD float		gDenoiseLerpC;
CD float		gNoIterations;
CD float		gInvNoIterations;

#define TF_NO_SAMPLES		128
#define INV_TF_NO_SAMPLES	1.0f / (float)TF_NO_SAMPLES

#include "Model.cuh"
#include "View.cuh"
#include "Blur.cuh"
#include "Denoise.cuh"
#include "Estimate.cuh"
#include "Utilities.cuh"
#include "SingleScattering.cuh"
#include "NearestIntersection.cuh"
#include "SpecularBloom.cuh"
#include "ToneMap.cuh"

CCudaModel	gModel;
CCudaView	gRenderCanvasView;
CCudaView	gNavigatorView;

void BindDensityBuffer(short* pBuffer, hipExtent Extent)
{
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<short>();

	HandleCudaError(hipMalloc3DArray(&gpDensityArray, &ChannelDesc, Extent));

	hipMemcpy3DParms CopyParams = {0};

	CopyParams.srcPtr	= make_hipPitchedPtr(pBuffer, Extent.width * sizeof(short), Extent.width, Extent.height);
	CopyParams.dstArray	= gpDensityArray;
	CopyParams.extent	= Extent;
	CopyParams.kind		= hipMemcpyHostToDevice;
	
	HandleCudaError(hipMemcpy3D(&CopyParams));

	gTexDensity.normalized		= true;
	gTexDensity.filterMode		= hipFilterModeLinear;      
	gTexDensity.addressMode[0]	= hipAddressModeClamp;  
	gTexDensity.addressMode[1]	= hipAddressModeClamp;
  	gTexDensity.addressMode[2]	= hipAddressModeClamp;

	HandleCudaError(hipBindTextureToArray(gTexDensity, gpDensityArray, ChannelDesc));
}

void BindGradientMagnitudeBuffer(short* pBuffer, hipExtent Extent)
{
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<short>();
	HandleCudaError(hipMalloc3DArray(&gpGradientMagnitudeArray, &ChannelDesc, Extent));

	hipMemcpy3DParms CopyParams = {0};

	CopyParams.srcPtr	= make_hipPitchedPtr(pBuffer, Extent.width * sizeof(short), Extent.width, Extent.height);
	CopyParams.dstArray	= gpGradientMagnitudeArray;
	CopyParams.extent	= Extent;
	CopyParams.kind		= hipMemcpyHostToDevice;
	
	HandleCudaError(hipMemcpy3D(&CopyParams));

	gTexGradientMagnitude.normalized		= true;
	gTexGradientMagnitude.filterMode		= hipFilterModeLinear;      
	gTexGradientMagnitude.addressMode[0]	= hipAddressModeClamp;  
	gTexGradientMagnitude.addressMode[1]	= hipAddressModeClamp;
  	gTexGradientMagnitude.addressMode[2]	= hipAddressModeClamp;

	HandleCudaError(hipBindTextureToArray(gTexGradientMagnitude, gpGradientMagnitudeArray, ChannelDesc));
}

void UnbindDensityBuffer(void)
{
	HandleCudaError(hipFreeArray(gpDensityArray));
	gpDensityArray = NULL;
	HandleCudaError(hipUnbindTexture(gTexDensity));
}

void UnbindGradientMagnitudeBuffer(void)
{
	HandleCudaError(hipFreeArray(gpGradientMagnitudeArray));
	gpGradientMagnitudeArray = NULL;
	HandleCudaError(hipUnbindTexture(gTexGradientMagnitude));
}

void BindRenderCanvasView(const CResolution2D& Resolution)
{
	gRenderCanvasView.Resize(Resolution);

	hipChannelFormatDesc Channel;
	
	Channel = hipCreateChannelDesc<uchar4>();

	HandleCudaError(hipBindTexture2D(0, gTexRunningEstimateRgba, gRenderCanvasView.m_EstimateRgbaLdr.GetPtr(), Channel, gRenderCanvasView.GetWidth(), gRenderCanvasView.GetHeight(), gRenderCanvasView.m_EstimateRgbaLdr.GetPitch()));
}

void ResetRenderCanvasView(void)
{
	gRenderCanvasView.Reset();
}

void FreeRenderCanvasView(void)
{
	gRenderCanvasView.Free();
}

unsigned char* GetDisplayEstimate(void)
{
	return (unsigned char*)gRenderCanvasView.m_DisplayEstimateRgbLdr.GetPtr(0, 0);
}

void BindTransferFunctionOpacity(CTransferFunction& TransferFunctionOpacity)
{
	gTexOpacity.normalized		= true;
	gTexOpacity.filterMode		= hipFilterModeLinear;
	gTexOpacity.addressMode[0]	= hipAddressModeClamp;

	float Opacity[TF_NO_SAMPLES];

	for (int i = 0; i < TF_NO_SAMPLES; i++)
		Opacity[i] = TransferFunctionOpacity.F((float)i * INV_TF_NO_SAMPLES).r;
	
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<float>();

	if (gpOpacityArray == NULL)
		HandleCudaError(hipMallocArray(&gpOpacityArray, &ChannelDesc, TF_NO_SAMPLES, 1));

	HandleCudaError(hipMemcpyToArray(gpOpacityArray, 0, 0, Opacity, TF_NO_SAMPLES * sizeof(float), hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexOpacity, gpOpacityArray, ChannelDesc));
}

void UnbindTransferFunctionOpacity(void)
{
	HandleCudaError(hipFreeArray(gpOpacityArray));
	gpOpacityArray = NULL;
	HandleCudaError(hipUnbindTexture(gTexOpacity));
}

void BindTransferFunctionDiffuse(CTransferFunction& TransferFunctionDiffuse)
{
	gTexDiffuse.normalized		= true;
	gTexDiffuse.filterMode		= hipFilterModeLinear;
	gTexDiffuse.addressMode[0]	= hipAddressModeClamp;

	float4 Diffuse[TF_NO_SAMPLES];

	for (int i = 0; i < TF_NO_SAMPLES; i++)
	{
		Diffuse[i].x = TransferFunctionDiffuse.F((float)i * INV_TF_NO_SAMPLES).r;
		Diffuse[i].y = TransferFunctionDiffuse.F((float)i * INV_TF_NO_SAMPLES).g;
		Diffuse[i].z = TransferFunctionDiffuse.F((float)i * INV_TF_NO_SAMPLES).b;
	}

	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<float4>();
	
	if (gpDiffuseArray == NULL)
		HandleCudaError(hipMallocArray(&gpDiffuseArray, &ChannelDesc, TF_NO_SAMPLES, 1));

	HandleCudaError(hipMemcpyToArray(gpDiffuseArray, 0, 0, Diffuse, TF_NO_SAMPLES * sizeof(float4), hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexDiffuse, gpDiffuseArray, ChannelDesc));
}

void UnbindTransferFunctionDiffuse(void)
{
	HandleCudaError(hipFreeArray(gpDiffuseArray));
	gpDiffuseArray = NULL;
	HandleCudaError(hipUnbindTexture(gTexDiffuse));
}

void BindTransferFunctionSpecular(CTransferFunction& TransferFunctionSpecular)
{
	gTexSpecular.normalized		= true;
	gTexSpecular.filterMode		= hipFilterModeLinear;
	gTexSpecular.addressMode[0]	= hipAddressModeClamp;

	float4 Specular[TF_NO_SAMPLES];

	for (int i = 0; i < TF_NO_SAMPLES; i++)
	{
		Specular[i].x = TransferFunctionSpecular.F((float)i * INV_TF_NO_SAMPLES).r;
		Specular[i].y = TransferFunctionSpecular.F((float)i * INV_TF_NO_SAMPLES).g;
		Specular[i].z = TransferFunctionSpecular.F((float)i * INV_TF_NO_SAMPLES).b;
	}

	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<float4>();
	
	if (gpSpecularArray == NULL)
		HandleCudaError(hipMallocArray(&gpSpecularArray, &ChannelDesc, TF_NO_SAMPLES, 1));

	HandleCudaError(hipMemcpyToArray(gpSpecularArray, 0, 0, Specular, TF_NO_SAMPLES * sizeof(float4), hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexSpecular, gpSpecularArray, ChannelDesc));
}

void UnbindTransferFunctionSpecular(void)
{
	HandleCudaError(hipFreeArray(gpSpecularArray));
	gpSpecularArray = NULL;
	HandleCudaError(hipUnbindTexture(gTexSpecular));
}

void BindTransferFunctionRoughness(CTransferFunction& TransferFunctionRoughness)
{
	gTexRoughness.normalized		= true;
	gTexRoughness.filterMode		= hipFilterModeLinear;
	gTexRoughness.addressMode[0]	= hipAddressModeClamp;

	float Roughness[TF_NO_SAMPLES];

	for (int i = 0; i < TF_NO_SAMPLES; i++)
		Roughness[i] = TransferFunctionRoughness.F((float)i * INV_TF_NO_SAMPLES).r;
	
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<float>();

	if (gpRoughnessArray == NULL)
		HandleCudaError(hipMallocArray(&gpRoughnessArray, &ChannelDesc, TF_NO_SAMPLES, 1));

	HandleCudaError(hipMemcpyToArray(gpRoughnessArray, 0, 0, Roughness, TF_NO_SAMPLES * sizeof(float),  hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexRoughness, gpRoughnessArray, ChannelDesc));
}

void UnbindTransferFunctionRoughness(void)
{
	HandleCudaError(hipFreeArray(gpRoughnessArray));
	gpRoughnessArray = NULL;
	HandleCudaError(hipUnbindTexture(gTexRoughness));
}

void BindTransferFunctionEmission(CTransferFunction& TransferFunctionEmission)
{
	gTexEmission.normalized		= true;
	gTexEmission.filterMode		= hipFilterModeLinear;
	gTexEmission.addressMode[0]	= hipAddressModeClamp;

	float4 Emission[TF_NO_SAMPLES];

	for (int i = 0; i < TF_NO_SAMPLES; i++)
	{
		Emission[i].x = TransferFunctionEmission.F((float)i * INV_TF_NO_SAMPLES).r;
		Emission[i].y = TransferFunctionEmission.F((float)i * INV_TF_NO_SAMPLES).g;
		Emission[i].z = TransferFunctionEmission.F((float)i * INV_TF_NO_SAMPLES).b;
	}

	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<float4>();
	
	if (gpEmissionArray == NULL)
		HandleCudaError(hipMallocArray(&gpEmissionArray, &ChannelDesc, TF_NO_SAMPLES, 1));

	HandleCudaError(hipMemcpyToArray(gpEmissionArray, 0, 0, Emission, TF_NO_SAMPLES * sizeof(float4),  hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexEmission, gpEmissionArray, ChannelDesc));
}

void UnbindTransferFunctionEmission(void)
{
	HandleCudaError(hipFreeArray(gpEmissionArray));
	gpEmissionArray = NULL;
	HandleCudaError(hipUnbindTexture(gTexEmission));
}

void BindConstants(CScene* pScene)
{
	const float3 AaBbMin = make_float3(pScene->m_BoundingBox.GetMinP().x, pScene->m_BoundingBox.GetMinP().y, pScene->m_BoundingBox.GetMinP().z);
	const float3 AaBbMax = make_float3(pScene->m_BoundingBox.GetMaxP().x, pScene->m_BoundingBox.GetMaxP().y, pScene->m_BoundingBox.GetMaxP().z);

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gAaBbMin), &AaBbMin, sizeof(float3)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gAaBbMax), &AaBbMax, sizeof(float3)));

	const float3 InvAaBbMin = make_float3(pScene->m_BoundingBox.GetInvMinP().x, pScene->m_BoundingBox.GetInvMinP().y, pScene->m_BoundingBox.GetInvMinP().z);
	const float3 InvAaBbMax = make_float3(pScene->m_BoundingBox.GetInvMaxP().x, pScene->m_BoundingBox.GetInvMaxP().y, pScene->m_BoundingBox.GetInvMaxP().z);

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gInvAaBbMin), &InvAaBbMin, sizeof(float3)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gInvAaBbMax), &InvAaBbMax, sizeof(float3)));

	const float IntensityMin		= pScene->m_IntensityRange.GetMin();
	const float IntensityMax		= pScene->m_IntensityRange.GetMax();
	const float IntensityRange		= pScene->m_IntensityRange.GetRange();
	const float IntensityInvRange	= 1.0f / IntensityRange;

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gIntensityMin), &IntensityMin, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gIntensityMax), &IntensityMax, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gIntensityRange), &IntensityRange, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gIntensityInvRange), &IntensityInvRange, sizeof(float)));

	const float StepSize		= pScene->m_StepSizeFactor * pScene->m_GradientDelta;
	const float StepSizeShadow	= pScene->m_StepSizeFactorShadow * pScene->m_GradientDelta;

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gStepSize), &StepSize, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gStepSizeShadow), &StepSizeShadow, sizeof(float)));

	const float DensityScale = pScene->m_DensityScale;

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gDensityScale), &DensityScale, sizeof(float)));
	
	const float GradientDelta		= 1.0f * pScene->m_GradientDelta;
	const float InvGradientDelta	= 1.0f / GradientDelta;
	const Vec3f GradientDeltaX(GradientDelta, 0.0f, 0.0f);
	const Vec3f GradientDeltaY(0.0f, GradientDelta, 0.0f);
	const Vec3f GradientDeltaZ(0.0f, 0.0f, GradientDelta);
	
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gGradientDelta), &GradientDelta, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gInvGradientDelta), &InvGradientDelta, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gGradientDeltaX), &GradientDeltaX, sizeof(Vec3f)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gGradientDeltaY), &GradientDeltaY, sizeof(Vec3f)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gGradientDeltaZ), &GradientDeltaZ, sizeof(Vec3f)));
	
	const int FilmWidth		= pScene->m_Camera.m_Film.GetWidth();
	const int Filmheight	= pScene->m_Camera.m_Film.GetHeight();
	const int FilmNoPixels	= pScene->m_Camera.m_Film.m_Resolution.GetNoElements();

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gFilmWidth), &FilmWidth, sizeof(int)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gFilmHeight), &Filmheight, sizeof(int)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gFilmNoPixels), &FilmNoPixels, sizeof(int)));

	const int FilterWidth = 1;

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gFilterWidth), &FilterWidth, sizeof(int)));

	const float FilterWeights[10] = { 0.11411459588254977f, 0.08176668094332218f, 0.03008028089187349f, 0.01f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f };

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gFilterWeights), &FilterWeights, 10 * sizeof(float)));

	const float Gamma		= pScene->m_Camera.m_Film.m_Gamma;
	const float InvGamma	= 1.0f / Gamma;
	const float Exposure	= pScene->m_Camera.m_Film.m_Exposure;
	const float InvExposure	= 1.0f / Exposure;

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gExposure), &Exposure, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gInvExposure), &InvExposure, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gGamma), &Gamma, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gInvGamma), &InvGamma, sizeof(float)));

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gDenoiseEnabled), &pScene->m_DenoiseParams.m_Enabled, sizeof(bool)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gDenoiseWindowRadius), &pScene->m_DenoiseParams.m_WindowRadius, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gDenoiseInvWindowArea), &pScene->m_DenoiseParams.m_InvWindowArea, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gDenoiseNoise), &pScene->m_DenoiseParams.m_Noise, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gDenoiseWeightThreshold), &pScene->m_DenoiseParams.m_WeightThreshold, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gDenoiseLerpThreshold), &pScene->m_DenoiseParams.m_LerpThreshold, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gDenoiseLerpC), &pScene->m_DenoiseParams.m_LerpC, sizeof(float)));

	const float NoIterations	= pScene->GetNoIterations();
	const float InvNoIterations = 1.0f / std::max(1.0f, NoIterations);

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gNoIterations), &NoIterations, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gInvNoIterations), &InvNoIterations, sizeof(float)));
}

void Render(const int& Type, CScene& Scene, CTiming& RenderImage, CTiming& BlurImage, CTiming& PostProcessImage, CTiming& DenoiseImage)
{
	CScene* pDevScene = NULL;

	HandleCudaError(hipMalloc(&pDevScene, sizeof(CScene)));
	HandleCudaError(hipMemcpy(pDevScene, &Scene, sizeof(CScene), hipMemcpyHostToDevice));

	if (Scene.m_Camera.m_Focus.m_Type == 0)
		Scene.m_Camera.m_Focus.m_FocalDistance = NearestIntersection(pDevScene);

	HandleCudaError(hipMemcpy(pDevScene, &Scene, sizeof(CScene), hipMemcpyHostToDevice));

	CCudaView* pDevView = NULL;

	HandleCudaError(hipMalloc(&pDevView, sizeof(CCudaView)));
	HandleCudaError(hipMemcpy(pDevView, &gRenderCanvasView, sizeof(CCudaView), hipMemcpyHostToDevice));

	
	CCudaTimer TmrRender;
	
	switch (Type)
	{
		case 0:
		{
			SingleScattering(&Scene, pDevScene, pDevView);
			break;
		}

		case 1:
		{
//			MultipleScattering(&Scene, pDevScene);
			break;
		}
	}

	RenderImage.AddDuration(TmrRender.ElapsedTime());
	
 	CCudaTimer TmrBlur;
	Blur(&Scene, pDevScene, pDevView);
	BlurImage.AddDuration(TmrBlur.ElapsedTime());

	CCudaTimer TmrPostProcess;
	Estimate(&Scene, pDevScene, pDevView);
	PostProcessImage.AddDuration(TmrPostProcess.ElapsedTime());

	ToneMap(&Scene, pDevScene, pDevView);

	CCudaTimer TmrDenoise;
	Denoise(&Scene, pDevScene, pDevView);
	DenoiseImage.AddDuration(TmrDenoise.ElapsedTime());
	

	HandleCudaError(hipFree(pDevScene));
	HandleCudaError(hipFree(pDevView));
}